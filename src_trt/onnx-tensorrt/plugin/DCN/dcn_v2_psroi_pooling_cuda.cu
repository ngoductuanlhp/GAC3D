#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2017 Microsoft
 * Licensed under The MIT License [see LICENSE for details]
 * \file deformable_psroi_pooling.cu
 * \brief
 * \author Yi Li, Guodong Zhang, Jifeng Dai
*/
/***************** Adapted by Charles Shang *********************/

#include <cstdio>
#include <algorithm>
#include <cstring>
#include <iostream>


#define CUDA_KERNEL_LOOP(i, n)                        \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n);                                       \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 512;
inline int GET_BLOCKS(const int N)
{
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename T>
__device__ T bilinear_interp(
    const T *data,
    const T x,
    const T y,
    const int width,
    const int height)
{
  int x1 = floor(x);
  int x2 = ceil(x);
  int y1 = floor(y);
  int y2 = ceil(y);
  T dist_x = static_cast<T>(x - x1);
  T dist_y = static_cast<T>(y - y1);
  T value11 = data[y1 * width + x1];
  T value12 = data[y2 * width + x1];
  T value21 = data[y1 * width + x2];
  T value22 = data[y2 * width + x2];
  T value = (1 - dist_x) * (1 - dist_y) * value11 +
            (1 - dist_x) * dist_y * value12 +
            dist_x * (1 - dist_y) * value21 +
            dist_x * dist_y * value22;
  return value;
}

template <typename T>
__global__ void DeformablePSROIPoolForwardKernel(
    const int count,
    const T *bottom_data,
    const T spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const T *bottom_rois, const T *bottom_trans,
    const int no_trans,
    const T trans_std,
    const int sample_per_part,
    const int output_dim,
    const int group_size,
    const int part_size,
    const int num_classes,
    const int channels_each_class,
    T *top_data,
    T *top_count)
{
  CUDA_KERNEL_LOOP(index, count)
  {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const T *offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    T roi_start_w = static_cast<T>(round(offset_bottom_rois[1])) * spatial_scale - 0.5;
    T roi_start_h = static_cast<T>(round(offset_bottom_rois[2])) * spatial_scale - 0.5;
    T roi_end_w = static_cast<T>(round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
    T roi_end_h = static_cast<T>(round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

    // Force too small ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
    T roi_height = max(roi_end_h - roi_start_h, 0.1);

    // Compute w and h at bottom
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    T sub_bin_size_h = bin_size_h / static_cast<T>(sample_per_part);
    T sub_bin_size_w = bin_size_w / static_cast<T>(sample_per_part);

    int part_h = floor(static_cast<T>(ph) / pooled_height * part_size);
    int part_w = floor(static_cast<T>(pw) / pooled_width * part_size);
    int class_id = ctop / channels_each_class;
    T trans_x = no_trans ? static_cast<T>(0) : bottom_trans[(((n * num_classes + class_id) * 2) * part_size + part_h) * part_size + part_w] * trans_std;
    T trans_y = no_trans ? static_cast<T>(0) : bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w] * trans_std;

    T wstart = static_cast<T>(pw) * bin_size_w + roi_start_w;
    wstart += trans_x * roi_width;
    T hstart = static_cast<T>(ph) * bin_size_h + roi_start_h;
    hstart += trans_y * roi_height;

    T sum = 0;
    int count = 0;
    int gw = floor(static_cast<T>(pw) * group_size / pooled_width);
    int gh = floor(static_cast<T>(ph) * group_size / pooled_height);
    gw = min(max(gw, 0), group_size - 1);
    gh = min(max(gh, 0), group_size - 1);

    const T *offset_bottom_data = bottom_data + (roi_batch_ind * channels) * height * width;
    for (int ih = 0; ih < sample_per_part; ih++)
    {
      for (int iw = 0; iw < sample_per_part; iw++)
      {
        T w = wstart + iw * sub_bin_size_w;
        T h = hstart + ih * sub_bin_size_h;
        // bilinear interpolation
        if (w < -0.5 || w > width - 0.5 || h < -0.5 || h > height - 0.5)
        {
          continue;
        }
        w = min(max(w, 0.), width - 1.);
        h = min(max(h, 0.), height - 1.);
        int c = (ctop * group_size + gh) * group_size + gw;
        T val = bilinear_interp(offset_bottom_data + c * height * width, w, h, width, height);
        sum += val;
        count++;
      }
    }
    top_data[index] = count == 0 ? static_cast<T>(0) : sum / count;
    top_count[index] = count;
  }
}

dcn_v2_psroi_pooling_cuda_forward(const at::Tensor &input,
                                  const at::Tensor &bbox,
                                  const at::Tensor &trans,
                                  const int no_trans,
                                  const float spatial_scale,
                                  const int output_dim,
                                  const int group_size,
                                  const int pooled_size,
                                  const int part_size,
                                  const int sample_per_part,
                                  const float trans_std)
{
  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(bbox.type().is_cuda(), "rois must be a CUDA tensor");
  AT_ASSERTM(trans.type().is_cuda(), "trans must be a CUDA tensor");

  const int batch = input.size(0);
  const int channels = input.size(1);
  const int height = input.size(2);
  const int width = input.size(3);
  const int channels_trans = no_trans ? 2 : trans.size(1);
  const int num_bbox = bbox.size(0);

  AT_ASSERTM(channels == output_dim, "input channels and output channels must equal");
  auto pooled_height = pooled_size;
  auto pooled_width = pooled_size;

  auto out = at::empty({num_bbox, output_dim, pooled_height, pooled_width}, input.options());
  long out_size = num_bbox * output_dim * pooled_height * pooled_width;
  auto top_count = at::zeros({num_bbox, output_dim, pooled_height, pooled_width}, input.options());

  const int num_classes = no_trans ? 1 : channels_trans / 2;
  const int channels_each_class = no_trans ? output_dim : output_dim / num_classes;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (out.numel() == 0)
  {
    THCudaCheck(hipGetLastError());
    return std::make_tuple(out, top_count);
  }

  dim3 grid(std::min(THCCeilDiv(out_size, 512L), 4096L));
  dim3 block(512);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "dcn_v2_psroi_pooling_cuda_forward", [&] {
    DeformablePSROIPoolForwardKernel<scalar_t><<<grid, block, 0, stream>>>(
        out_size,
        input.contiguous().data<scalar_t>(),
        spatial_scale,
        channels,
        height, width,
        pooled_height,
        pooled_width,
        bbox.contiguous().data<scalar_t>(),
        trans.contiguous().data<scalar_t>(),
        no_trans,
        trans_std,
        sample_per_part,
        output_dim,
        group_size,
        part_size,
        num_classes,
        channels_each_class,
        out.data<scalar_t>(),
        top_count.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return std::make_tuple(out, top_count);
}

